#include "../include/datastructs.h"
extern "C" {
#include <lofs-dirstruct.h>
#include <lofs-hdf2nc.h>
#include <lofs-limits.h>
#include <lofs-macros.h>
}
#include <iostream>
#ifndef DATASTRUCTS
#define DATASTRUCTS
/*
 * Copyright (C) 2017-2020 Kelton Halbert, Space Science and Engineering Center (SSEC), University of Wisconsin - Madison
 * Written by Kelton Halbert at the University of Wisconsin - Madison,
 * Cooperative Institute for Meteorological Satellite Studies (CIMSS),
 * Space Science and Engineering Center (SSEC). Provided under the Apache 2.0 License.
 * Email: kthalbert@wisc.edu
*/
using namespace std;

/* Allocate memory on the CPU and GPU for a grid. There are times,
    like for various MPI ranks, that you don't want to do this on both.
    See the similar function for doing this on just the CPU */
mesh* allocate_mesh_managed( hdf_meta *hm, grid *gd ) { 
    mesh *msh;

    hipMallocManaged(&msh, sizeof(mesh));


    // allocage grid arrays
    hipMallocManaged(&(msh->xffull), (hm->nx+1)*sizeof(float));
    hipMallocManaged(&(msh->xhfull), (hm->nx)*sizeof(float));
    hipMallocManaged(&(msh->xfout), (gd->NX)*sizeof(float));
    hipMallocManaged(&(msh->xhout), (gd->NX)*sizeof(float));

    hipMallocManaged(&(msh->yffull), (hm->ny+1)*sizeof(float));
    hipMallocManaged(&(msh->yffull), (hm->ny)*sizeof(float));
    hipMallocManaged(&(msh->yfout), (gd->NY)*sizeof(float));
    hipMallocManaged(&(msh->yhout), (gd->NY)*sizeof(float));

    // +2 is +1 for stagger, +1 for potential bottom ghost zone
    hipMallocManaged(&(msh->zf), (gd->NZ)*sizeof(float));
    hipMallocManaged(&(msh->zh), (gd->NZ)*sizeof(float));
    hipMallocManaged(&(msh->zfout), (gd->NZ)*sizeof(float));
    hipMallocManaged(&(msh->zhout), (gd->NZ)*sizeof(float));

    hipMallocManaged(&(msh->uf), (gd->NX+2)*sizeof(float));
    hipMallocManaged(&(msh->uh), (gd->NX+2)*sizeof(float));

    hipMallocManaged(&(msh->vf), (gd->NY+2)*sizeof(float));
    hipMallocManaged(&(msh->vh), (gd->NY+2)*sizeof(float));

    hipMallocManaged(&(msh->mf), (gd->NZ+2)*sizeof(float));
    hipMallocManaged(&(msh->mh), (gd->NZ+2)*sizeof(float));
    return msh;
}

/* Allocate arrays only on the CPU for the grid. This is important
   for using with MPI, as only 1 rank should be allocating memory
   on the GPU */
mesh* allocate_mesh_cpu( hdf_meta *hm, grid *gd ) { 
    mesh *msh = new mesh();

    // allocage grid arrays
    msh->xffull = new float[hm->nx+1];
    msh->xhfull = new float[hm->nx];
    msh->xfout = new float[gd->NX];
    msh->xhout = new float[gd->NX];

    msh->yffull = new float[hm->ny+1];
    msh->yhfull = new float[hm->ny];
    msh->yfout = new float[gd->NY];
    msh->yhout = new float[gd->NY];

    msh->zfout = new float[gd->NZ];
    msh->zhout = new float[gd->NZ];
    msh->zf = new float[gd->NZ];
    msh->zh = new float[gd->NZ];

    msh->uf = new float[gd->NX+2];
    msh->uh = new float[gd->NX+2];

    msh->vf = new float[gd->NY+2];
    msh->vh = new float[gd->NY+2];

    msh->mf = new float[gd->NZ+2];
    msh->mh = new float[gd->NZ+2];
	return msh;
}

sounding* allocate_sounding_managed(grid *gd) {
	sounding *snd;
    hipMallocManaged(&snd, sizeof(sounding));
    // allocate base state arrays
    hipMallocManaged(&(snd->u0),   (gd->NZ)*sizeof(float));
    hipMallocManaged(&(snd->v0),   (gd->NZ)*sizeof(float));
    hipMallocManaged(&(snd->qv0),  (gd->NZ)*sizeof(float));
    hipMallocManaged(&(snd->th0),  (gd->NZ)*sizeof(float));
    hipMallocManaged(&(snd->rho0), (gd->NZ)*sizeof(float));
    hipMallocManaged(&(snd->pres0),   (gd->NZ)*sizeof(float));
	return snd;
}

sounding* allocate_sounding_cpu(grid *gd) {

	sounding *snd = new sounding();
    // allocate base state arrays
    snd->u0 = new float[gd->NZ];
    snd->v0 = new float[gd->NZ];
    snd->qv0 = new float[gd->NZ];
    snd->th0 = new float[gd->NZ];
    snd->rho0 = new float[gd->NZ];
    snd->pres0 = new float[gd->NZ];

    return snd;
}

/* Deallocate all of the arrays in the 
   struct for both the GPU and CPU */
void deallocate_mesh_managed(mesh *msh) {
    hipFree(msh->xffull);
    hipFree(msh->xhfull);
    hipFree(msh->xfout);
    hipFree(msh->xhout);
    hipFree(msh->yffull);
    hipFree(msh->yhfull);
    hipFree(msh->yfout);
    hipFree(msh->yhout);
    hipFree(msh->zfout);
    hipFree(msh->zhout);
    hipFree(msh->zf);
    hipFree(msh->zh);
    hipFree(msh->uf);
    hipFree(msh->uh);
    hipFree(msh->vf);
    hipFree(msh->vh);
    hipFree(msh->mf);
    hipFree(msh->mh);
}

/* Deallocate all of the arrays in the
   struct only for the CPU */
void deallocate_mesh_cpu(mesh *msh) {
    delete[] msh->xffull;
    delete[] msh->xhfull;
    delete[] msh->xfout;
    delete[] msh->xhout;
    delete[] msh->yffull;
    delete[] msh->yhfull;
    delete[] msh->yfout;
    delete[] msh->yhout;
    delete[] msh->zfout;
    delete[] msh->zhout;
    delete[] msh->zf;
    delete[] msh->zh;
    delete[] msh->uf;
    delete[] msh->uh;
    delete[] msh->vf;
    delete[] msh->vh;
    delete[] msh->mf;
    delete[] msh->mh;
}

void deallocate_sounding_managed(sounding *snd) {
    hipFree(snd->u0);
    hipFree(snd->v0);
    hipFree(snd->rho0);
    hipFree(snd->th0);
    hipFree(snd->qv0);
    hipFree(snd->pres0);
}

void deallocate_sounding_cpu(sounding *snd) {
    delete[] snd->u0;
    delete[] snd->v0;
    delete[] snd->rho0;
    delete[] snd->th0;
    delete[] snd->qv0;
    delete[] snd->pres0;
}

/* Allocate arrays for parcel info on both the CPU and GPU.
   This function should only be called by MPI Rank 0, so
   be sure to use the CPU function for Rank >= 1. */
parcel_pos* allocate_parcels_managed(iocfg *io, int NX, int NY, int NZ, int nTotTimes) {
    int nParcels = NX*NY*NZ;
    parcel_pos *parcels;
    // create the struct on both the GPU and the CPU.
    hipMallocManaged(&parcels, sizeof(parcel_pos));
    hipMallocManaged(&(parcels->io), sizeof(iocfg));
    // set the values of the struct on the GPU
    parcels->io->output_pbar = io->output_pbar; 
    parcels->io->output_qvbar = io->output_qvbar;
    parcels->io->output_rhobar = io->output_rhobar;
    parcels->io->output_thetabar = io->output_thetabar;
    parcels->io->output_thrhobar = io->output_thrhobar;

    parcels->io->output_ppert = io->output_ppert;
    parcels->io->output_qvpert = io->output_qvpert;
    parcels->io->output_rhopert = io->output_rhopert;
    parcels->io->output_thetapert = io->output_thetapert;
    parcels->io->output_thrhopert = io->output_thrhopert;

    parcels->io->output_qc = io->output_qc;
    parcels->io->output_qi = io->output_qi;
    parcels->io->output_qs = io->output_qs;
    parcels->io->output_qg = io->output_qg;

    parcels->io->output_xvort = io->output_xvort;
    parcels->io->output_yvort = io->output_yvort;
    parcels->io->output_zvort = io->output_zvort;

    parcels->io->output_kmh = io->output_kmh;

    parcels->io->output_vorticity_budget = io->output_vorticity_budget;
    parcels->io->output_momentum_budget = io->output_momentum_budget;
    
    // allocate memory for the parcels
    // we are integrating for the entirety 
    // of the simulation.
    hipMallocManaged(&(parcels->xpos), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->ypos), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->zpos), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclu), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclv), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclw), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_kmh ) hipMallocManaged(&(parcels->pclkmh), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_momentum_budget) {
        hipMallocManaged(&(parcels->pclbuoy), nParcels*nTotTimes*sizeof(float));
        hipMallocManaged(&(parcels->pclupgrad), nParcels*nTotTimes*sizeof(float));
        hipMallocManaged(&(parcels->pclvpgrad), nParcels*nTotTimes*sizeof(float));
        hipMallocManaged(&(parcels->pclwpgrad), nParcels*nTotTimes*sizeof(float));
        hipMallocManaged(&(parcels->pcluturb), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclvturb), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclwturb), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pcludiff), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclvdiff), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclwdiff), nParcels*nTotTimes*sizeof(float)); 
    }
    if (io->output_xvort || io->output_vorticity_budget) hipMallocManaged(&(parcels->pclxvort), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_yvort || io->output_vorticity_budget) hipMallocManaged(&(parcels->pclyvort), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_zvort || io->output_vorticity_budget) hipMallocManaged(&(parcels->pclzvort), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_vorticity_budget) {
        hipMallocManaged(&(parcels->pclxvorttilt), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclyvorttilt), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclzvorttilt), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclxvortstretch), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclyvortstretch), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclzvortstretch), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclxvortturb), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclyvortturb), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclzvortturb), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclxvortdiff), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclyvortdiff), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclzvortdiff), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclxvortbaro), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclyvortbaro), nParcels*nTotTimes*sizeof(float)); 
        hipMallocManaged(&(parcels->pclxvortsolenoid), nParcels*nTotTimes*sizeof(float));
        hipMallocManaged(&(parcels->pclyvortsolenoid), nParcels*nTotTimes*sizeof(float));
        hipMallocManaged(&(parcels->pclzvortsolenoid), nParcels*nTotTimes*sizeof(float));
    }

    if (io->output_ppert) hipMallocManaged(&(parcels->pclppert), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_qvpert) hipMallocManaged(&(parcels->pclqvpert), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_rhopert) hipMallocManaged(&(parcels->pclrhopert), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_thetapert) hipMallocManaged(&(parcels->pclthetapert), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_thrhopert) hipMallocManaged(&(parcels->pclthrhopert), nParcels*nTotTimes*sizeof(float)); 

    if (io->output_pbar) hipMallocManaged(&(parcels->pclpbar), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_qvbar) hipMallocManaged(&(parcels->pclqvbar), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_rhobar) hipMallocManaged(&(parcels->pclrhobar), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_thetabar) hipMallocManaged(&(parcels->pclthetabar), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_thrhobar) hipMallocManaged(&(parcels->pclthrhobar), nParcels*nTotTimes*sizeof(float)); 

    if (io->output_qc) hipMallocManaged(&(parcels->pclqc), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_qi) hipMallocManaged(&(parcels->pclqi), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_qs) hipMallocManaged(&(parcels->pclqs), nParcels*nTotTimes*sizeof(float)); 
    if (io->output_qg) hipMallocManaged(&(parcels->pclqg), nParcels*nTotTimes*sizeof(float)); 

    // set the static variables
    parcels->nParcels = nParcels;
    parcels->nTimes = nTotTimes;
    hipDeviceSynchronize();

    return parcels;
}

/* Allocate arrays only on the CPU for the grid. This is important
   for using with MPI, as only 1 rank should be allocating memory
   on the GPU */
parcel_pos* allocate_parcels_cpu(iocfg* io, int NX, int NY, int NZ, int nTotTimes) {
    int nParcels = NX*NY*NZ;
    parcel_pos *parcels = new parcel_pos();
    parcels->io = io;

    // allocate memory for the parcels
    // we are integrating for the entirety 
    // of the simulation.
    parcels->xpos = new float[nParcels*nTotTimes]; 
    parcels->ypos = new float[nParcels*nTotTimes]; 
    parcels->zpos = new float[nParcels*nTotTimes]; 
    parcels->pclu = new float[nParcels*nTotTimes]; 
    parcels->pclv = new float[nParcels*nTotTimes]; 
    parcels->pclw = new float[nParcels*nTotTimes]; 
    if (io->output_kmh) parcels->pclkmh = new float[nParcels*nTotTimes]; 
    if (io->output_momentum_budget) {
        parcels->pclbuoy = new float[nParcels*nTotTimes];
        parcels->pclupgrad = new float[nParcels*nTotTimes];
        parcels->pclvpgrad = new float[nParcels*nTotTimes];
        parcels->pclwpgrad = new float[nParcels*nTotTimes];
        parcels->pcluturb = new float[nParcels*nTotTimes]; 
        parcels->pclvturb = new float[nParcels*nTotTimes]; 
        parcels->pclwturb = new float[nParcels*nTotTimes]; 
        parcels->pcludiff = new float[nParcels*nTotTimes]; 
        parcels->pclvdiff = new float[nParcels*nTotTimes]; 
        parcels->pclwdiff = new float[nParcels*nTotTimes]; 
    }
    if (io->output_vorticity_budget || io->output_xvort) parcels->pclxvort = new float[nParcels*nTotTimes]; 
    if (io->output_vorticity_budget || io->output_yvort) parcels->pclyvort = new float[nParcels*nTotTimes]; 
    if (io->output_vorticity_budget || io->output_zvort) parcels->pclzvort = new float[nParcels*nTotTimes]; 
    if (io->output_vorticity_budget) {
        parcels->pclxvorttilt = new float[nParcels*nTotTimes]; 
        parcels->pclyvorttilt = new float[nParcels*nTotTimes]; 
        parcels->pclzvorttilt = new float[nParcels*nTotTimes]; 
        parcels->pclxvortstretch = new float[nParcels*nTotTimes]; 
        parcels->pclyvortstretch = new float[nParcels*nTotTimes]; 
        parcels->pclzvortstretch = new float[nParcels*nTotTimes]; 
        parcels->pclxvortturb = new float[nParcels*nTotTimes]; 
        parcels->pclyvortturb = new float[nParcels*nTotTimes]; 
        parcels->pclzvortturb = new float[nParcels*nTotTimes]; 
        parcels->pclxvortdiff = new float[nParcels*nTotTimes]; 
        parcels->pclyvortdiff = new float[nParcels*nTotTimes]; 
        parcels->pclzvortdiff = new float[nParcels*nTotTimes]; 
        parcels->pclxvortbaro = new float[nParcels*nTotTimes]; 
        parcels->pclyvortbaro = new float[nParcels*nTotTimes]; 
        parcels->pclxvortsolenoid = new float[nParcels*nTotTimes];
        parcels->pclyvortsolenoid = new float[nParcels*nTotTimes];
        parcels->pclzvortsolenoid = new float[nParcels*nTotTimes];
    }

    if (io->output_ppert) parcels->pclppert = new float[nParcels*nTotTimes];
    if (io->output_qvpert) parcels->pclqvpert = new float[nParcels*nTotTimes];
    if (io->output_rhopert) parcels->pclrhopert = new float[nParcels*nTotTimes];
    if (io->output_thetapert) parcels->pclthetapert = new float[nParcels*nTotTimes];
    if (io->output_thrhopert) parcels->pclthrhopert = new float[nParcels*nTotTimes];

    if (io->output_pbar) parcels->pclpbar = new float[nParcels*nTotTimes];
    if (io->output_qvbar) parcels->pclqvbar = new float[nParcels*nTotTimes];
    if (io->output_rhobar) parcels->pclrhobar = new float[nParcels*nTotTimes];
    if (io->output_thetabar) parcels->pclthetabar = new float[nParcels*nTotTimes];
    if (io->output_thrhobar) parcels->pclthrhobar = new float[nParcels*nTotTimes];

    if (io->output_qc) parcels->pclqc = new float[nParcels*nTotTimes];
    if (io->output_qi) parcels->pclqi = new float[nParcels*nTotTimes];
    if (io->output_qs) parcels->pclqs = new float[nParcels*nTotTimes];
    if (io->output_qg) parcels->pclqg = new float[nParcels*nTotTimes];
    // set the static variables
    parcels->nParcels = nParcels;
    parcels->nTimes = nTotTimes;

    return parcels;
}

/* Deallocate parcel arrays on both the CPU and the
   GPU */
void deallocate_parcels_managed(iocfg* io, parcel_pos *parcels) {
    hipFree(parcels->xpos);
    hipFree(parcels->ypos);
    hipFree(parcels->zpos);
    hipFree(parcels->pclu);
    hipFree(parcels->pclv);
    hipFree(parcels->pclw);
    if (io->output_kmh) hipFree(parcels->pclkmh);
    if (io->output_momentum_budget) {
        hipFree(parcels->pclbuoy);
        hipFree(parcels->pclupgrad);
        hipFree(parcels->pclvpgrad);
        hipFree(parcels->pclwpgrad);
        hipFree(parcels->pcluturb);
        hipFree(parcels->pclvturb);
        hipFree(parcels->pclwturb);
        hipFree(parcels->pcludiff);
        hipFree(parcels->pclvdiff);
        hipFree(parcels->pclwdiff);
    }
    if (io->output_vorticity_budget || io->output_xvort) hipFree(parcels->pclxvort);
    if (io->output_vorticity_budget || io->output_yvort) hipFree(parcels->pclyvort);
    if (io->output_vorticity_budget || io->output_zvort) hipFree(parcels->pclzvort);
    if (io->output_vorticity_budget) {
        hipFree(parcels->pclxvorttilt);
        hipFree(parcels->pclyvorttilt);
        hipFree(parcels->pclzvorttilt);
        hipFree(parcels->pclxvortstretch);
        hipFree(parcels->pclyvortstretch);
        hipFree(parcels->pclzvortstretch);
        hipFree(parcels->pclxvortturb);
        hipFree(parcels->pclyvortturb);
        hipFree(parcels->pclzvortturb);
        hipFree(parcels->pclxvortdiff);
        hipFree(parcels->pclyvortdiff);
        hipFree(parcels->pclzvortdiff);
        hipFree(parcels->pclxvortbaro);
        hipFree(parcels->pclyvortbaro);
        hipFree(parcels->pclxvortsolenoid);
        hipFree(parcels->pclyvortsolenoid);
        hipFree(parcels->pclzvortsolenoid);
    }

    if (io->output_ppert) hipFree(parcels->pclppert);
    if (io->output_qvpert) hipFree(parcels->pclqvpert);
    if (io->output_rhopert) hipFree(parcels->pclrhopert);
    if (io->output_thetapert) hipFree(parcels->pclthetapert);
    if (io->output_rhopert) hipFree(parcels->pclthrhopert);

    if (io->output_pbar) hipFree(parcels->pclpbar);
    if (io->output_qvbar) hipFree(parcels->pclqvbar);
    if (io->output_rhobar) hipFree(parcels->pclrhobar);
    if (io->output_thetabar) hipFree(parcels->pclthetabar);
    if (io->output_thrhobar) hipFree(parcels->pclthrhobar);

    if (io->output_qc) hipFree(parcels->pclqc);
    if (io->output_qi) hipFree(parcels->pclqi);
    if (io->output_qs) hipFree(parcels->pclqs);
    if (io->output_qg) hipFree(parcels->pclqg);

    hipFree(parcels);
    hipDeviceSynchronize();
}

/* Deallocate parcel arrays only on the CPU */
void deallocate_parcels_cpu(iocfg *io, parcel_pos *parcels) {
    delete[] parcels->xpos;
    delete[] parcels->ypos;
    delete[] parcels->zpos;
    delete[] parcels->pclu;
    delete[] parcels->pclv;
    delete[] parcels->pclw;
    if (io->output_kmh) delete[] parcels->pclkmh;
    if (io->output_momentum_budget) {
        delete[] parcels->pclbuoy;
        delete[] parcels->pclupgrad;
        delete[] parcels->pclvpgrad;
        delete[] parcels->pclwpgrad;
        delete[] parcels->pcluturb;
        delete[] parcels->pclvturb;
        delete[] parcels->pclwturb;
        delete[] parcels->pcludiff;
        delete[] parcels->pclvdiff;
        delete[] parcels->pclwdiff;
    }
    if (io->output_vorticity_budget || io->output_xvort) delete[] parcels->pclxvort;
    if (io->output_vorticity_budget || io->output_yvort) delete[] parcels->pclyvort;
    if (io->output_vorticity_budget || io->output_zvort) delete[] parcels->pclzvort;
    if (io->output_vorticity_budget) {
        delete[] parcels->pclxvorttilt;
        delete[] parcels->pclyvorttilt;
        delete[] parcels->pclzvorttilt;
        delete[] parcels->pclxvortstretch;
        delete[] parcels->pclyvortstretch;
        delete[] parcels->pclzvortstretch;
        delete[] parcels->pclxvortturb;
        delete[] parcels->pclyvortturb;
        delete[] parcels->pclzvortturb;
        delete[] parcels->pclxvortdiff;
        delete[] parcels->pclyvortdiff;
        delete[] parcels->pclzvortdiff;
        delete[] parcels->pclxvortbaro;
        delete[] parcels->pclyvortbaro;
        delete[] parcels->pclxvortsolenoid;
        delete[] parcels->pclyvortsolenoid;
        delete[] parcels->pclzvortsolenoid;
    }

    if (io->output_ppert) delete[] parcels->pclppert;
    if (io->output_qvpert) delete[] parcels->pclqvpert;
    if (io->output_rhopert) delete[] parcels->pclrhopert;
    if (io->output_thetapert) delete[] parcels->pclthetapert;
    if (io->output_thrhopert) delete[] parcels->pclthrhopert;

    if (io->output_pbar) delete[] parcels->pclpbar;
    if (io->output_qvbar) delete[] parcels->pclqvbar;
    if (io->output_rhobar) delete[] parcels->pclrhobar;
    if (io->output_thetabar) delete[] parcels->pclthetabar;
    if (io->output_thrhobar) delete[] parcels->pclthrhobar;

    if (io->output_qc) delete[] parcels->pclqc;
    if (io->output_qi) delete[] parcels->pclqi;
    if (io->output_qs) delete[] parcels->pclqs;
    if (io->output_qg) delete[] parcels->pclqg;

    delete[] parcels;
}

/* Allocate the struct of 4D arrays that store
   fields for integration and calculation. This
   only ever gets called by Rank 0, so there 
   should be no need for a CPU counterpart. */
model_data* allocate_model_managed(iocfg *io, long bufsize) {
    model_data *data;
    // create the struct on both the GPU and the CPU.
    hipMallocManaged(&data, sizeof(model_data));
    hipMallocManaged(&(data->io), sizeof(iocfg));
    // set the values of the struct on the GPU
    data->io->output_pbar = io->output_pbar; 
    data->io->output_qvbar = io->output_qvbar;
    data->io->output_rhobar = io->output_rhobar;
    data->io->output_thetabar = io->output_thetabar;
    data->io->output_thrhobar = io->output_thrhobar;

    data->io->output_ppert = io->output_ppert;
    data->io->output_qvpert = io->output_qvpert;
    data->io->output_rhopert = io->output_rhopert;
    data->io->output_thetapert = io->output_thetapert;
    data->io->output_thrhopert = io->output_thrhopert;

    data->io->output_qc = io->output_qc;
    data->io->output_qi = io->output_qi;
    data->io->output_qs = io->output_qs;
    data->io->output_qg = io->output_qg;

    data->io->output_xvort = io->output_xvort;
    data->io->output_yvort = io->output_yvort;
    data->io->output_zvort = io->output_zvort;

    data->io->output_kmh = io->output_kmh;

    data->io->output_vorticity_budget = io->output_vorticity_budget;
    data->io->output_momentum_budget = io->output_momentum_budget;

    // Now, here we only allocate the arrays that we need based on the
    // user supplied namelist configuration. This should help with a)
    // not having to manually comment out the microphysics variables
    // every time, and b) save on memory load when possible. 

    // These are arrays that are 100% necessary for parcel integration.
    // The temporary arrays are included in this because pretty much any
    // secondary calculation requires at least one or more of these
    // arrays. So, better to just have them up front. 
    hipMallocManaged(&(data->ustag), bufsize*sizeof(float));
    hipMallocManaged(&(data->vstag), bufsize*sizeof(float));
    hipMallocManaged(&(data->wstag), bufsize*sizeof(float));
    hipMallocManaged(&(data->tem1), bufsize*sizeof(float));
    hipMallocManaged(&(data->tem2), bufsize*sizeof(float));
    hipMallocManaged(&(data->tem3), bufsize*sizeof(float));
    hipMallocManaged(&(data->tem4), bufsize*sizeof(float));
    hipMallocManaged(&(data->tem5), bufsize*sizeof(float));
    hipMallocManaged(&(data->tem6), bufsize*sizeof(float));
    
    // Arrays that are optional depending on if they need to be tracked along
    // a parcel, or are part of a calculation/budget. 
    if (io->output_qc) hipMallocManaged(&(data->qc), bufsize*sizeof(float));
    if (io->output_qi) hipMallocManaged(&(data->qi), bufsize*sizeof(float));
    if (io->output_qs) hipMallocManaged(&(data->qs), bufsize*sizeof(float));
    if (io->output_qg) hipMallocManaged(&(data->qg), bufsize*sizeof(float));

    if (io->output_vorticity_budget || io->output_xvort) hipMallocManaged(&(data->xvort), bufsize*sizeof(float));
    if (io->output_vorticity_budget || io->output_yvort) hipMallocManaged(&(data->yvort), bufsize*sizeof(float));
    if (io->output_vorticity_budget || io->output_zvort) hipMallocManaged(&(data->zvort), bufsize*sizeof(float));

    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_ppert) hipMallocManaged(&(data->pipert), bufsize*sizeof(float));
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_ppert) hipMallocManaged(&(data->prespert), bufsize*sizeof(float));
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_thrhopert) hipMallocManaged(&(data->thrhopert),  bufsize*sizeof(float));
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_thetapert) hipMallocManaged(&(data->thetapert),  bufsize*sizeof(float));
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_rhopert) hipMallocManaged(&(data->rhopert), bufsize*sizeof(float));
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_kmh) hipMallocManaged(&(data->kmh), bufsize*sizeof(float));
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_qvpert) hipMallocManaged(&(data->qvpert), bufsize*sizeof(float));
    if (io->output_vorticity_budget || io->output_momentum_budget) {
        hipMallocManaged(&(data->rhof), bufsize*sizeof(float));
        hipMallocManaged(&(data->buoy), bufsize*sizeof(float));
        hipMallocManaged(&(data->pgradu), bufsize*sizeof(float));
        hipMallocManaged(&(data->pgradv), bufsize*sizeof(float));
        hipMallocManaged(&(data->pgradw), bufsize*sizeof(float));
        hipMallocManaged(&(data->turbu), bufsize*sizeof(float));
        hipMallocManaged(&(data->turbv), bufsize*sizeof(float));
        hipMallocManaged(&(data->turbw), bufsize*sizeof(float));
        hipMallocManaged(&(data->diffu), bufsize*sizeof(float));
        hipMallocManaged(&(data->diffv), bufsize*sizeof(float));
        hipMallocManaged(&(data->diffw), bufsize*sizeof(float));
    }
    if (io->output_vorticity_budget) {
        hipMallocManaged(&(data->xvtilt), bufsize*sizeof(float));
        hipMallocManaged(&(data->yvtilt), bufsize*sizeof(float));
        hipMallocManaged(&(data->zvtilt), bufsize*sizeof(float));
        hipMallocManaged(&(data->xvstretch), bufsize*sizeof(float));
        hipMallocManaged(&(data->yvstretch), bufsize*sizeof(float));
        hipMallocManaged(&(data->zvstretch), bufsize*sizeof(float));
        hipMallocManaged(&(data->turbxvort), bufsize*sizeof(float));
        hipMallocManaged(&(data->turbyvort), bufsize*sizeof(float));
        hipMallocManaged(&(data->turbzvort), bufsize*sizeof(float));
        hipMallocManaged(&(data->diffxvort), bufsize*sizeof(float));
        hipMallocManaged(&(data->diffyvort), bufsize*sizeof(float));
        hipMallocManaged(&(data->diffzvort), bufsize*sizeof(float));
        hipMallocManaged(&(data->xvort_baro), bufsize*sizeof(float)); 
        hipMallocManaged(&(data->yvort_baro), bufsize*sizeof(float)); 
        hipMallocManaged(&(data->xvort_solenoid), bufsize*sizeof(float)); 
        hipMallocManaged(&(data->yvort_solenoid), bufsize*sizeof(float)); 
        hipMallocManaged(&(data->zvort_solenoid), bufsize*sizeof(float)); 
    }

    return data;

}

/* Deallocate the struct of 4D arrays that store
   fields for integration and calculation. This 
   only ever gets called by Rank 0, so there
   should be no need for a CPU counterpart. */
void deallocate_model_managed(iocfg *io, model_data *data) {
    hipFree(data->ustag);
    hipFree(data->vstag);
    hipFree(data->wstag);
    hipFree(data->tem1);
    hipFree(data->tem2);
    hipFree(data->tem3);
    hipFree(data->tem4);
    hipFree(data->tem5);
    hipFree(data->tem6);

    if (io->output_qc) hipFree(data->qc);
    if (io->output_qi) hipFree(data->qi);
    if (io->output_qs) hipFree(data->qs);
    if (io->output_qg) hipFree(data->qg);

    if (io->output_vorticity_budget || io->output_xvort) hipFree(data->xvort);
    if (io->output_vorticity_budget || io->output_yvort) hipFree(data->yvort);
    if (io->output_vorticity_budget || io->output_zvort) hipFree(data->zvort);

    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_ppert) hipFree(data->pipert);
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_ppert) hipFree(data->prespert);
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_thrhopert) hipFree(data->thrhopert);
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_thetapert) hipFree(data->thetapert);
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_rhopert) hipFree(data->rhopert);
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_kmh) hipFree(data->kmh);
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_qvpert) hipFree(data->qvpert);
    if (io->output_vorticity_budget || io->output_momentum_budget) {
        hipFree(data->rhof);
        hipFree(data->buoy);
        hipFree(data->pgradu);
        hipFree(data->pgradv);
        hipFree(data->pgradw);
        hipFree(data->turbu);
        hipFree(data->turbv);
        hipFree(data->turbw);
        hipFree(data->diffu);
        hipFree(data->diffv);
        hipFree(data->diffw);
    }
    if (io->output_vorticity_budget) {
        hipFree(data->xvtilt);
        hipFree(data->yvtilt);
        hipFree(data->zvtilt);
        hipFree(data->xvstretch);
        hipFree(data->yvstretch);
        hipFree(data->zvstretch);
        hipFree(data->turbxvort);
        hipFree(data->turbyvort);
        hipFree(data->turbzvort);
        hipFree(data->diffxvort);
        hipFree(data->diffyvort);
        hipFree(data->diffzvort);
        hipFree(data->xvort_baro);
        hipFree(data->yvort_baro);
        hipFree(data->xvort_solenoid); 
        hipFree(data->yvort_solenoid); 
        hipFree(data->zvort_solenoid); 
    }
}
#endif
