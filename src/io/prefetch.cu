
extern "C" {
#include <lofs-read.h>
#include <lofs-dirstruct.h>
#include <lofs-hdf2nc.h>
#include <lofs-limits.h>
#include <lofs-macros.h>
}

#include "../include/datastructs.h"
#include "../include/prefetch.h"
using namespace std;

#ifndef PREFETCH 
#define PREFETCH 
void prefetch_parcels_cpu(iocfg *io, parcel_pos *parcels, hipStream_t memStream) {
    int nParcels = parcels->nParcels;
    int nTotTimes = parcels->nTimes;
    hipMemPrefetchAsync(parcels->xpos, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    hipMemPrefetchAsync(parcels->ypos, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    hipMemPrefetchAsync(parcels->zpos, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    hipMemPrefetchAsync(parcels->pclu, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    hipMemPrefetchAsync(parcels->pclv, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    hipMemPrefetchAsync(parcels->pclw, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_kmh ) hipMemPrefetchAsync(parcels->pclkmh, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_momentum_budget) {
        hipMemPrefetchAsync(parcels->pclbuoy, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream);
        hipMemPrefetchAsync(parcels->pclupgrad, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream);
        hipMemPrefetchAsync(parcels->pclvpgrad, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream);
        hipMemPrefetchAsync(parcels->pclwpgrad, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream);
        hipMemPrefetchAsync(parcels->pcluturb, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclvturb, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclwturb, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pcludiff, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclvdiff, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclwdiff, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    }
    if (io->output_xvort || io->output_vorticity_budget) hipMemPrefetchAsync(parcels->pclxvort, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_yvort || io->output_vorticity_budget) hipMemPrefetchAsync(parcels->pclyvort, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_zvort || io->output_vorticity_budget) hipMemPrefetchAsync(parcels->pclzvort, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_vorticity_budget) {
        hipMemPrefetchAsync(parcels->pclxvorttilt, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclyvorttilt, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclzvorttilt, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclxvortstretch, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclyvortstretch, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclzvortstretch, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclxvortturb, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclyvortturb, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclzvortturb, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclxvortdiff, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclyvortdiff, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclzvortdiff, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclxvortbaro, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclyvortbaro, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
        hipMemPrefetchAsync(parcels->pclxvortsolenoid, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream);
        hipMemPrefetchAsync(parcels->pclyvortsolenoid, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream);
        hipMemPrefetchAsync(parcels->pclzvortsolenoid, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream);
    }

    if (io->output_ppert) hipMemPrefetchAsync(parcels->pclppert, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_qvpert) hipMemPrefetchAsync(parcels->pclqvpert, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_rhopert) hipMemPrefetchAsync(parcels->pclrhopert, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_thetapert) hipMemPrefetchAsync(parcels->pclthetapert, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_thrhopert) hipMemPrefetchAsync(parcels->pclthrhopert, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 

    if (io->output_pbar) hipMemPrefetchAsync(parcels->pclpbar, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_qvbar) hipMemPrefetchAsync(parcels->pclqvbar, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_rhobar) hipMemPrefetchAsync(parcels->pclrhobar, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_thetabar) hipMemPrefetchAsync(parcels->pclthetabar, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_thrhobar) hipMemPrefetchAsync(parcels->pclthrhobar, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 

    if (io->output_qc) hipMemPrefetchAsync(parcels->pclqc, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_qi) hipMemPrefetchAsync(parcels->pclqi, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_qs) hipMemPrefetchAsync(parcels->pclqs, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
    if (io->output_qg) hipMemPrefetchAsync(parcels->pclqg, nParcels*nTotTimes*sizeof(float), hipCpuDeviceId, memStream); 
}

void prefetch_model_gpu(iocfg *io, model_data *data, long bufsize, hipStream_t memStream) {
	int device = -1;
	hipGetDevice(&device);

    hipMemPrefetchAsync(data->ustag, bufsize*sizeof(float), device, memStream);
    hipMemPrefetchAsync(data->vstag, bufsize*sizeof(float), device, memStream);
    hipMemPrefetchAsync(data->wstag, bufsize*sizeof(float), device, memStream);
    
    // Arrays that are optional depending on if they need to be tracked along
    // a parcel, or are part of a calculation/budget. 
    if (io->output_qc) hipMemPrefetchAsync(data->qc, bufsize*sizeof(float), device, memStream);
    if (io->output_qi) hipMemPrefetchAsync(data->qi, bufsize*sizeof(float), device, memStream);
    if (io->output_qs) hipMemPrefetchAsync(data->qs, bufsize*sizeof(float), device, memStream);
    if (io->output_qg) hipMemPrefetchAsync(data->qg, bufsize*sizeof(float), device, memStream);


    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_ppert) hipMemPrefetchAsync(data->pipert, bufsize*sizeof(float), device, memStream);
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_ppert) hipMemPrefetchAsync(data->prespert, bufsize*sizeof(float), device, memStream);
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_thrhopert) hipMemPrefetchAsync(data->thrhopert,  bufsize*sizeof(float), device, memStream);
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_thetapert) hipMemPrefetchAsync(data->thetapert,  bufsize*sizeof(float), device, memStream);
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_rhopert) hipMemPrefetchAsync(data->rhopert, bufsize*sizeof(float), device, memStream);
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_kmh) hipMemPrefetchAsync(data->kmh, bufsize*sizeof(float), device, memStream);
    if (io->output_vorticity_budget || io->output_momentum_budget || io->output_qvpert) hipMemPrefetchAsync(data->qvpert, bufsize*sizeof(float), device, memStream);
}
#endif
